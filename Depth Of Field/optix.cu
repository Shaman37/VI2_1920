#include "hip/hip_runtime.h"

#include <optix.h>
#include "random.h"
#include "LaunchParams7.h" // our launch params
#include <vec_math.h> // NVIDIAs math utils

#define 	M_PI_2f   1.57079632679489661923f
#define 	M_PI_4f   0.78539816339744830962f

extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}
//  a single ray type
enum { PHONG=0, SHADOW, RAY_TYPE_COUNT };

struct colorPRD{
    float3 color;
    unsigned int seed;
} ;

struct shadowPRD{
    float shadowAtt;
    unsigned int seed;
} ;

// -------------------------------------------------------
// closest hit computes color based lolely on the triangle normal
// -------------------------------------------------------
static __device__ __inline__ float2 concentric_sampling(const float2& uOffset)
{

	// Handle degeneracy at origin
	if (uOffset.x == 0 && uOffset.y == 0)
		return make_float2(0.f, 0.f);

	// Apply concentric mapping to point
	float theta;
	float r;
	if(abs(uOffset.x) > abs(uOffset.y))
	{
		r = uOffset.x;
		theta = M_PI_4f * (uOffset.y / uOffset.x);
	}
	else
	{
		r = uOffset.y;
		theta = M_PI_2f - M_PI_4f * (uOffset.x / uOffset.y);
	}


	return r * make_float2(cos(theta), sin(theta));
}

static __device__ __inline__ float2 disc_sampling(const float2& u,
	const float maxwidth,
	const float maxheight,
	const uchar2& index)
{

	float w = (-(1 / 2) + ((index.x + 0.5) / maxwidth));
		
	float h = ((1 / 2) - ((index.y + 0.5) / maxheight));

	// a) Map uniform random number to [-1,1]^2
	float2 uOffset = 2.f * u - make_float2(1.f, 1.f);

	float2 image_point = make_float2(w, h);

	image_point += uOffset;

	return concentric_sampling(image_point);
}

extern "C" __global__ void __closesthit__radiance() {

    float3 &prd = *(float3*)getPRD<float3>();

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    // intersection position
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();

    // direction towards light
    float3 lPos = make_float3(optixLaunchParams.global->lightPos);
    float lDirLength = length(lPos - pos) - 0.01f;
    float3 lDir = normalize(lPos - pos);
    float3 nn = normalize(make_float3(n));

    float intensity = max(dot(lDir, nn),0.0f);

    // ray payload
    float shadowAttPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &shadowAttPRD, u0, u1 );  
  
    // trace shadow ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        lDir,
        0.001f,         // tmin
        lDirLength,     // tmax
        0.0f,           // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1 );

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {  
        // get barycentric coordinates
        // compute pixel texture coordinate
        const float4 tc
          = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
          +         u * sbtData.vertexD.texCoord0[index.y]
          +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
        prd = make_float3(fromTexture) * min(intensity * shadowAttPRD + 0.0, 1.0);
    }
    else
        prd = sbtData.color * min(intensity * shadowAttPRD + 0.0, 1.0);
}


// any hit to ignore intersections with back facing geometry
extern "C" __global__ void __anyhit__radiance() {

}

// miss sets the background color
extern "C" __global__ void __miss__radiance() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}

// -------------
// Shadow rays
// -------------

extern "C" __global__ void __closesthit__shadow() {

    float &prd = *(float*)getPRD<float>();
    prd = 0.0f;
}

// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {

}

// miss for shadows
extern "C" __global__ void __miss__shadow() {

    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}

// -----------------------------------------------
// Light material


extern "C" __global__ void __closesthit__light() {

    float3 &prd = *(float3*)getPRD<float3>();
    prd = make_float3(1.0f, 1.0f, 1.0f);
}


extern "C" __global__ void __anyhit__light() {
}


extern "C" __global__ void __miss__light() {
}


extern "C" __global__ void __closesthit__light_shadow() {

    float &prd = *(float*)getPRD<float>();
    prd = 1.0f;
}


// any hit to ignore intersections based on alpha transparency
extern "C" __global__ void __anyhit__light_shadow() {
}


// miss sets the background color
extern "C" __global__ void __miss__light_shadow() {
}




// -----------------------------------------------
// Metal Phong rays

extern "C" __global__ void __closesthit__phong_metal() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];
    // ray payload

    float3 normal = normalize(make_float3(n));

    // entering glass
    //if (dot(optixGetWorldRayDirection(), normal) < 0)

    float3 afterPRD = make_float3(1.0f);
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    //(1.f-u-v) * A + u * B + v * C;
    
    float3 rayDir = reflect(optixGetWorldRayDirection(), normal);
    optixTrace(optixLaunchParams.traversable,
        pos,
        rayDir,
        0.04f,    // tmin is high to void self-intersection
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        PHONG,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        PHONG,             // missSBTIndex 
        u0, u1 );

    float3 &prd = *(float3*)getPRD<float3>();
    prd = make_float3(0.8,0.8,0.8) * afterPRD;
}





// -----------------------------------------------
// Glass Phong rays

SUTIL_INLINE SUTIL_HOSTDEVICE float3 refract(const float3& i, const float3& n, const float eta) {

    float k = 1.0 - eta * eta * (1.0 - dot(n, i) * dot(n, i));
    if (k < 0.0)
        return make_float3(0.0f);
    else
        return (eta * i - (eta * dot(n, i) + sqrt(k)) * n);
}


extern "C" __global__ void __closesthit__phong_glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 normal = normalize(make_float3(n));
    const float3 normRayDir = optixGetWorldRayDirection();

    // new ray direction
    float3 rayDir;
    // entering glass
    float dotP;
    if (dot(normRayDir, normal) < 0) {
        dotP = dot(normRayDir, -normal);
        rayDir = refract(normRayDir, normal, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        rayDir = refract(normRayDir, -normal, 1.5);
    }

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    
    float3 refractPRD = make_float3(0.0f);
    uint32_t u0, u1;
    packPointer( &refractPRD, u0, u1 );  
    
    if (length(rayDir) > 0)
        optixTrace(optixLaunchParams.traversable,
            pos,
            rayDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );

    // ray payload 
    float3 &prd = *(float3*)getPRD<float3>();
 
    float3 reflectPRD = make_float3(0.0f);
    if (dotP > 0) {
        float3 reflectDir = reflect(normRayDir, normal);        
        packPointer( &reflectPRD, u0, u1 );  
        optixTrace(optixLaunchParams.traversable,
            pos,
            reflectDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );
        float r0 = (1.5f - 1.0f)/(1.5f + 1.0f);
        r0 = r0*r0 + (1-r0*r0) * pow(1-dotP,5);
        prd =  refractPRD * (1-r0) + r0*reflectPRD;
    }
    else
        prd =  refractPRD ;
}



extern "C" __global__ void __anyhit__phong_glass() {

}


// miss sets the background color
extern "C" __global__ void __miss__phong_glass() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}



// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {

    // ray payload
    float afterPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        optixGetWorldRayDirection(),
        0.001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1 );

    float &prd = *(float*)getPRD<float>();
    prd = 0.95f * afterPRD;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow_glass() {

    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}

// --------------
// Primary Rays
// --------------

extern "C" __global__ void __raygen__renderFrame() {

    //-- compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  

    //-- ray payload// ray payload
    colorPRD pixelColorPRD;
    pixelColorPRD.color = make_float3(1.f);

    //-- ray's direction computation
    const float2 screen(make_float2(ix+.5f,iy+.5f)/ make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);

    float3 ray_dir = normalize(camera.direction
       + screen.x  * camera.horizontal
       + screen.y * camera.vertical);

    //-- N ray samples
    float raysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    
    //-- color values (RGB)
    float red = 0.0f, green = 0.0f, blue = 0.0f;
     
    //-- Lens values calculation 
    float aperture = optixLaunchParams.global->aperture;
    float focal_length = optixLaunchParams.global->focalDistance;

    //-- if 'aperture == 0' treat camera as a 'pinhole camera'
    if(aperture == 0)
	{   
        
        uint32_t u0, u1;
        packPointer( &pixelColorPRD, u0, u1 );

           // trace primary ray
        optixTrace(optixLaunchParams.traversable,
            camera.position + camera.direction * optixLaunchParams.global->lensDistance,
            ray_dir,
            0.f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE,//,OPTIX_RAY_FLAG_DISABLE_ANYHIT
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,               // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );

            red = pixelColorPRD.color.x ;
            green = pixelColorPRD.color.y ;
            blue = pixelColorPRD.color.z ;
    }
    else{
        //-- Step 1 -> Calculate distance to the 'Focus Plane'
		float ft = focal_length / dot(ray_dir,camera.direction);
        float3 focus_plane = camera.position + ray_dir * ft;
        
        for (int i = 0; i < raysPerPixel; ++i) {
            for (int j = 0; j < raysPerPixel; ++j) {
    
                uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, i*raysPerPixel + j );
                
                //-- Step 2 -> Calculate sample point on lens (using Concentric Disk Sampling)
                float2 p_sample = make_float2(rnd(seed), rnd(seed));
				float2 disc_sample = disc_sampling(p_sample,raysPerPixel,raysPerPixel,make_uchar2(i,j));
                float2 lens_point = aperture * disc_sample;
                
                //-- Step 3 -> Compute point 'P' on 'Focus Plane'
				float3 ray_origin = camera.position + lens_point.x * camera.horizontal + lens_point.y * camera.vertical;
                float3 ray_direction = normalize(focus_plane - ray_origin);
                
                pixelColorPRD.seed = seed;
                uint32_t u0, u1;
                packPointer( &pixelColorPRD, u0, u1 );
                
                //-- Step 4 -> Trace ray's emitted from point 'P'            
                optixTrace(optixLaunchParams.traversable,
                        ray_origin + camera.direction * optixLaunchParams.global->lensDistance,
                        ray_direction,
                        0.f,    // tmin
                        1e20f,  // tmax
                        0.0f,   // rayTime
                        OptixVisibilityMask( 255 ),
                        OPTIX_RAY_FLAG_NONE,//,OPTIX_RAY_FLAG_DISABLE_ANYHIT
                        PHONG,             // SBT offset
                        RAY_TYPE_COUNT,               // SBT stride
                        PHONG,             // missSBTIndex 
                        u0, u1 );
    
                red += pixelColorPRD.color.x / (raysPerPixel*raysPerPixel);
                green += pixelColorPRD.color.y / (raysPerPixel*raysPerPixel);
                blue += pixelColorPRD.color.z / (raysPerPixel*raysPerPixel);
            }
        }
    }

    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*red);
    const int g = int(255.0f*green);
    const int b = int(255.0f*blue);
    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}
  

